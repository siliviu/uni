#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cassert>
#include <chrono>
#include <utility>
#include <hip/hip_runtime.h>

constexpr int TILE_WIDTH = 10;
constexpr int BLOCK_SIZE = 10;
constexpr float EPS = 1e-2;

// https://alvinwan.com/how-to-tile-matrix-multiplication/

__global__ void matrixMultiplyTiled(float* d_A, float* d_B, float* d_C, int N) {
    __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tile_B[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float value = 0;

    for (int k = 0; k < (N + TILE_WIDTH - 1) / TILE_WIDTH; k++) {
        if (row < N && k * TILE_WIDTH + threadIdx.x < N)
            tile_A[threadIdx.y][threadIdx.x] = d_A[row * N + k * TILE_WIDTH + threadIdx.x];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0;

        if (col < N && k * TILE_WIDTH + threadIdx.y < N)
            tile_B[threadIdx.y][threadIdx.x] = d_B[(k * TILE_WIDTH + threadIdx.y) * N + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++)
            value += tile_A[threadIdx.y][i] * tile_B[i][threadIdx.x];

        __syncthreads();
    }

    if (row < N && col < N)
        d_C[row * N + col] = value;
}

__global__ void matrixMultiply(float* d_A, float* d_B, float* d_C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0;

        for (int k = 0; k < N; ++k) {
            value += d_A[row * N + k] * d_B[k * N + col];
        }

        d_C[row * N + col] = value;
    }
}


void matrixMultiplyCUDATiled(float* h_A, float* h_B, float* h_C, int N) {
    size_t bytes = N * N * sizeof(float);

    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 dimGrid((N + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH, 1);

    matrixMultiplyTiled << <dimGrid, dimBlock >> > (d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void matrixMultiplyCUDA(float* h_A, float* h_B, float* h_C, int N) {
    size_t bytes = N * N * sizeof(float);

    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);


    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);

    matrixMultiply << <gridDim, blockDim >> > (d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}


void matrixMultiplyCPU(float* h_A, float* h_B, float* h_C, int N) {
    for (int i = 0;i < N;++i)
        for (int j = 0;j < N;++j) {
            float res = 0;
            for (int k = 0;k < N;++k)
                res += h_A[i * N + k] * h_B[k * N + j];
            h_C[i * N + j] = res;
        }
}

int main(int argc, char** argv) {
    assert(argc == 2);
    int N = atoi(argv[1]);

    float* h_A = new float[N * N];
    float* h_B = new float[N * N];
    float* h_C1 = new float[N * N];
    float* h_C2 = new float[N * N];
    float* h_C0 = new float[N * N];

    for (int i = 0; i < N * N; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    auto t_start = std::chrono::high_resolution_clock::now();
    matrixMultiplyCUDA(h_A, h_B, h_C1, N);
    auto t_end = std::chrono::high_resolution_clock::now();
    double elapsed_time_ms = std::chrono::duration<double, std::nano>(t_end - t_start).count();
    std::cout << "CUDA" << std::endl;
    std::cout << std::setprecision(10) << elapsed_time_ms << "\n";

    t_start = std::chrono::high_resolution_clock::now();
    matrixMultiplyCUDATiled(h_A, h_B, h_C2, N);
    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::nano>(t_end - t_start).count();
    std::cout << "CUDA Optimised" << std::endl;
    std::cout << std::setprecision(10) << elapsed_time_ms << "\n";

    t_start = std::chrono::high_resolution_clock::now();
    matrixMultiplyCPU(h_A, h_B, h_C0, N);
    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::nano>(t_end - t_start).count();
    std::cout << "CPU" << std::endl;
    std::cout << std::setprecision(10) << elapsed_time_ms << "\n";

    for (int i = 0;i < N * N;++i)
        assert(abs(h_C0[i] - h_C1[i]) <= EPS);

    for (int i = 0;i < N * N;++i)
        assert(abs(h_C0[i] - h_C2[i]) <= EPS);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C0;
    delete[] h_C1;
    delete[] h_C2;
}
